#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <stdio.h>

using namespace thrust;
using namespace std;

#define W 1000 //liczba w�tkow
#define K 100 //liczba generowanych punkt�w przez kazdy w�tek

#pragma region handle randoms

__global__ void __launch_bounds__(1024, 2) initRand(unsigned long long seed, hiprandState_t *state, int n)
{
	unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
	if (tid < n) {
		hiprand_init(seed, tid, 0, &state[tid]);
	}
}

__global__ void computeRandom(hiprandState_t *state, double *tab, int n)
{
	unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;

	if (tid<n) {
		hiprandState_t st = state[tid];

		//przerabiam int na double
		tab[tid] = (double)hiprand(&st) / 10000.0;
		tab[tid] -= (int)tab[tid];

		state[tid] = st;
	}
}

device_vector<double> generateRands()
{
	unsigned long int count = W * K * 2; //2*K liczb dla W w�tk�w
	unsigned int bs = 1024; //Rozmiar bloku
	unsigned long long int seed = 12346;
	device_vector<hiprandState_t> d_states; //stany generatora  liczb prseudolosowych dla ka�dego w�tku
	device_vector<double> randoms; //wynikowe liczby

	d_states.resize(count);
	randoms.resize(count);

	dim3 grid = dim3(ceil((double)count / (double)bs));

	initRand << <grid, bs >> > (seed, d_states.data().get(), d_states.size());
	computeRandom << <grid, bs >> > (d_states.data().get(), randoms.data().get(), randoms.size());
	hipDeviceSynchronize();

	return randoms;
}

#pragma endregion

__global__ void kernelCount(int *a, double *randoms, int w, int k)
{
	int threadId = threadIdx.x;

	int s = 0;

	if (threadId < w)
	{
		for (int i = 0; i < k; i++)
		{
			double x = randoms[threadId * i * 2];
			double y = randoms[threadId * i * 2 + 1];

			if ((x * x) + (y * y) < 1)
				s++;
		}

		a[threadId] = s;
	}
}

unsigned long sumArray(int *a, int num_elements)
{
	int i = 0;
	unsigned long sum = 0;
	for (i = 0; i<num_elements; i++)
		sum = sum + a[i];

	return(sum);
}

int main()
{
	auto rands = generateRands();
	cout << "Liczby losowe wygenerowano pomyslnie" << endl;

	/*for (int i = 0; i < rands.size(); i++) {
		cout << "rands[" << i << "] = " << rands[i] << endl;
	}*/

	int *A;
	hipMallocManaged(&A, W * sizeof(int));

	kernelCount <<< 1, W >>> (A, rands.data().get(), W, K);

	hipDeviceSynchronize();

	/*for (int i = 0; i < W; i++) {
		cout << "a[" << i << "] = " << A[i] << endl;
	}*/

	unsigned long sum = sumArray(A, W);
	double pi = ((double)sum * 4.0) / (double)(W * K);

	cout << "PI = " << pi << endl;

	hipFree(A);

	return 0;
}